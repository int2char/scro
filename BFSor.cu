#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void BFSor::copydata(int s,vector<edge>&edges,int nodenum){
};
void BFSor::dellocate(){
};
void BFSor::allocate(int maxn,int maxedge){
}
void BFSor::topsort()
{
};
__global__ void cleanb(int *d,int *p,int N,int numoff)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=N)return;
	d[i+numoff]=100000;
	p[i+numoff]=-1;
};
/*__global__ void cleanb(int *d,int *p,int N)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=N)return;
	d[i]=100000;
	p[i]=-1;
};*/
void BFSor::updatE(vector<vector<int>>&tesigns)
{
	esigns=tesigns;
	int cou1=0;
	for(int k=0;k<LY;k++)
	{
		for(int i=0;i<pnodesize;i++)
		for(int j=0;j<rus[i].size();j++)
			rudw[cou1++]=esigns[k][ruw[i][j]];
	}
	hipMemcpy(dev_rudw,rudw,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
};
__global__ void Sorb(int *d,int *p,int *sor,int ly,int ye,int yoff,int numoff)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=ly)return;
	int l=i/ye;
	int id=i%ye;
	int y=sor[id+yoff];
	d[y*ly+l*ye+id+numoff]=0;
};
void BFSor::updatS(vector<vector<Sot>>&stpair)
{
	L[0]=0;
	L[1]=LY1;
	L[2]=LY2;
	S[0]=stpair[0].size();
	S[1]=stpair[1].size();
	stps=stpair;
	ncount=L[1]*S[0]+L[2]*S[1];
	int bigN=ncount*pnodesize;
	int numoff=L[1]*S[0]*pnodesize;
		
	int count=0;
	for(int j=0;j<stpair[0].size();j++)
		sor[count++]=stpair[0][j].s;
	int fs=count;
	for(int j=0;j<stpair[1].size();j++)
		sor[count++]=stpair[1][j].s;
	hipMemcpy(dev_sor,sor,count*sizeof(int),hipMemcpyHostToDevice);
	cleanb<<<L[1]*S[0]*pnodesize/512+1,512>>>(dev_d,dev_p,L[1]*S[0]*pnodesize,0);
	cleanb<<<L[2]*S[1]*pnodesize/512+1,512>>>(dev_d,dev_p,L[2]*S[1]*pnodesize,numoff);
	Sorb<<<L[1]*S[0]/512+1,512>>>(dev_d,dev_p,dev_sor,L[1]*S[0],S[0],0,0);
	Sorb<<<L[2]*S[1]/512+1,512>>>(dev_d,dev_p,dev_sor,L[2]*S[1],S[1],fs,numoff);
	Size[0]=nodenum*L[1]*S[0];
	Size[1]=nodenum*L[2]*S[1];
}
void BFSor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,int _nodenum)
{
	//cout<<"in paraller BFS init"<<endl;
	nodenum=_nodenum;
	pnodesize=nodenum;
	edges=ext.first;
	esigns=ext.second;
	stp=stpair;
	W=WD+1;
	//st=new int[edges.size()*LY];
	//te=new int[edges.size()*LY];
	d=new int[nodenum*LY*YE];
	p=new int[nodenum*LY*YE];
	esignes=new int[edges.size()*LY];
	vector<vector<int>>neibn(pnodesize*LY,vector<int>());
	vector<vector<int>>neie(pnodesize,vector<int>());
	vector<vector<int>>rs(pnodesize,vector<int>());
	vector<vector<int>>rw(pnodesize,vector<int>());
	rus=rs;
	ruw=rw;
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			rus[t].push_back(s);
			ruw[t].push_back(i);
			neibn[s].push_back(t);
			neie[s].push_back(i);
		}
	rudu=new int[edges.size()];
	rudw=new int[edges.size()*LY];
	rid=new int[edges.size()];
	int cou1=0;
	int cou2=0;
	int cou3=0;
	mm=new int[pnodesize+1];
	ss=new int[pnodesize+1];
	int du=0;
	for(int i=0;i<pnodesize;i++)
		{
			ss[i]=rus[i].size();
			mm[i]=du;
			du+=rus[i].size();
			for(int j=0;j<rus[i].size();j++)
				rudu[cou1++]=rus[i][j];
			for(int j=0;j<rus[i].size();j++)	
				rid[cou3++]=ruw[i][j];
		}
	for(int k=0;k<LY;k++)
	{
		for(int i=0;i<pnodesize;i++)
		for(int j=0;j<rus[i].size();j++)
			rudw[cou2++]=esigns[k][ruw[i][j]];
	}
	int count=0;
	sor=new int[2*YE];

	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_p,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_mm,(pnodesize+1)*sizeof(int));
	hipMalloc((void**)&dev_ss,(pnodesize+1)*sizeof(int));
	hipMalloc((void**)&dev_rudu,edges.size()*sizeof(int));
	hipMalloc((void**)&dev_rudw,edges.size()*LY*sizeof(int));
	hipMalloc((void**)&dev_rid,edges.size()*sizeof(int));
	hipMalloc((void**)&dev_sor,2*YE*sizeof(int));
	hipMemcpy(dev_rudu,rudu,edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rudw,rudw,edges.size()*LY*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rid,rid,edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_mm,mm,(pnodesize+1)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_ss,ss,(pnodesize+1)*sizeof(int),hipMemcpyHostToDevice);
};
BFSor::BFSor():L(PC+1,0),S(PC,0),NF(PC,0),Size(2,0)
{
};
__global__ void BFSFu(int *rudu,int*rudw,int *rid,int *d,int*p,int K,int EE,int PN,int sizeoff,int leveloff,int yel,int ye,int*mm,int *ss)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=yel)return;
	int ly=i/ye+leveloff;
	int nid=blockIdx.y;
	int off=sizeoff;
	int ii=nid*yel+i+sizeoff;
	int dm=d[ii];
	for(int k=mm[nid];k<ss[nid]+mm[nid];k++)
		{
			int node=rudu[k];
			if(rudw[k+EE*ly]<0)continue;
			int v=d[off+node*yel+i];
			if(v==K-1&&dm>v){d[ii]=K;break;}
		}
}
__global__ void BFScolor(int *rudu,int*rudw,int *rid,int *d,int*p,int K,int EE,int PN,int sizeoff,int leveloff,int yel,int ye,int*mm,int *ss)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=yel)return;
	int ly=i/ye+leveloff;
	int nid=blockIdx.y;
	int off=sizeoff;
	int ii=nid*yel+i+sizeoff;
	int dm=d[ii];
	for(int k=mm[nid];k<ss[nid]+mm[nid];k++)
		{
			int node=rudu[k];
			if(rudw[k+EE*ly]<0)continue;
			int v=d[off+node*yel+i];
			if(v+1==d[ii]){p[ii]=rid[k];break;}
		}
}
/*__global__ void BFSfast(int *st,int *te,int *d,int* p,int *stid,int E,int N,int size,int round,int Leveloff,int numoff,int yel,int ye)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=yel)return;	
	int eid=blockIdx.y;
	int eeid=(i/ye+Leveloff)*E+eid;
	int s=st[eeid],t=te[eeid];
	if(t<0)return;
	int offs=s*yel+numoff;
	int offt=t*yel+numoff;
	if(d[offs+i]==round-1&&d[offt+i]>round)d[offt+i]=round;
}
__global__ void BFScolor(int *st,int *te,int *d,int* p,int *stid,int E,int N,int size,int round,int Leveloff,int numoff,int yel,int ye)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=yel)return;	
	int eid=blockIdx.y;
	int eeid=(i/ye+Leveloff)*E+eid;
	int s=st[eeid],t=te[eeid];
	if(t<0)return;
	int offs=s*yel+numoff;
	int offt=t*yel+numoff;
	if(d[offs+i]==d[offt+i]-1)p[offt+i]=stid[eeid];
}*/
vector<vector<Rout>> BFSor::routalg(int s,int t,int bw)
{
	//cout<<"blasting "<<endl;
	int kk=1;
	time_t start,end;
	start=clock();
	int size=edges.size()*LY*YE;
	hipStream_t stream0;
	hipStreamCreate(&stream0);
	hipStream_t stream1;
	hipStreamCreate(&stream1);
	int leoff=L[1];
	int nuoff=L[1]*S[0]*nodenum;
	dim3 blocks_s1(S[0]*L[1]/512+1,pnodesize);
	dim3 blocks_s2(S[1]*L[2]/512+1,pnodesize);
	int sizeoff=S[0]*L[1]*nodenum;

	for(int i=1;i<WD+1;i++)
		{
			BFSFu<<<blocks_s1,512,0,stream0>>>(dev_rudu,dev_rudw,dev_rid,dev_d,dev_p,i,edges.size(),pnodesize,0,0,S[0]*L[1],S[0],dev_mm,dev_ss);
			BFSFu<<<blocks_s2,512,0,stream1>>>(dev_rudu,dev_rudw,dev_rid,dev_d,dev_p,i,edges.size(),pnodesize,sizeoff,L[1],S[1]*L[2],S[1],dev_mm,dev_ss);
		}
	BFScolor<<<blocks_s1,512,0,stream0>>>(dev_rudu,dev_rudw,dev_rid,dev_d,dev_p,0,edges.size(),pnodesize,0,0,S[0]*L[1],S[0],dev_mm,dev_ss);
	BFScolor<<<blocks_s2,512,0,stream1>>>(dev_rudu,dev_rudw,dev_rid,dev_d,dev_p,0,edges.size(),pnodesize,sizeoff,L[1],S[1]*L[2],S[1],dev_mm,dev_ss);
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream0);
	hipMemcpy(d,dev_d,ncount*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(p,dev_p,ncount*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	/*for(int i=0;i<1;i++)
	{
		cout<<"********************************************** "<<i<<endl;
		for(int j=3;j<4;j++)
			{
					cout<<endl;
					for(int g=0;g<pnodesize;g++)
						{
							//cout<<p[g*S[0]*L[1]+i*S[0]+j]<<" ";
						}
				}
	}*/
	vector<vector<Rout>>result(2,vector<Rout>());
	vector<int>LL(3,0);
	LL=L;
	LL[2]+=LL[1];
	int count=0;
	int offg=0;
	for(int y=1;y<PC+1;y++)
		{
		int teoff=S[y-1]*L[y];
		for(int k=LL[y-1];k<LL[y];k++)
		{	
			if(y==2)offg=sizeoff;
			int boff=(k-LL[y-1])*S[y-1]+offg;
			for(int l=0;l<stps[y-1].size();l++)
			{	int loff=boff+l;
				int s=stps[y-1][l].s;
				vector<int>ters=stps[y-1][l].ters;
				for(int i=0;i<ters.size();i++)
				{
					int id=stps[y-1][l].mmpid[ters[i]];
					int hop=0;
					int tt=ters[i];
					int min=d[tt*teoff+loff];
					if(min>50000)continue;
					int of=loff;
					Rout S(s,tt,id,min,of,k);
					result[y-1].push_back(S);
				}
				count++;
			}
		}
		}
	end=clock();
	//cout<<"GPU time is : "<<end-start<<endl;
	//cout<<"over!"<<endl;
	//hipFree(dev_te);
	//hipFree(dev_st);
	//hipFree(dev_d);
	//cout<<"before return"<<endl;
	return result;
};
/*__global__ void BFSfast(int *st,int *te,int *d,int round,int E,int N,int size)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;
	int eid=(i%(E*LY));
	int s=st[eid],t=te[eid];
	int off=(i/(E*LY))*N+(eid/E)*N*YE;
	if(d[s+off]==round-1&&d[t+off]>round)
		d[t+off]=round;
}*/
