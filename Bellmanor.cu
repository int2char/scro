#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void Bellmanor::copydata(int s,vector<edge>&edges,int nodenum){
};
void Bellmanor::dellocate(){
};
void Bellmanor::allocate(int maxn,int maxedge){
}
void Bellmanor::topsort()
{
};
void Bellmanor::updatE(vector<vector<int>>&tesigns)
{
	esigns=tesigns;
	int cou1=0;
	for(int k=0;k<LY;k++)
	{
		for(int i=0;i<pnodesize;i++)
		for(int j=0;j<rus[i].size();j++)
			rudw[cou1++]=esigns[k][ruw[i][j]];
	}
	hipMemcpy(dev_rudw,rudw,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
}

__global__ void clean(int *d,int *p,int N)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=N)return;
	d[i]=100000;
	p[i]=-1;
};
void Bellmanor::updatS(vector<vector<Sot>>&stpair)
{
	L[0]=0;
	L[1]=LY1;
	L[2]=LY2;
	S[0]=stpair[0].size();
	S[1]=stpair[1].size();
	stps=stpair;
	int count=0;
	ncount=L[1]*S[0]+L[2]*S[1];
	int bigN=ncount*nodenum;
	clean<<<bigN/512+1,512,0>>>(dev_d,dev_p,bigN);
	hipMemcpy(d,dev_d,ncount*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	for(int k=0;k<L[1];k++)
		{
		for(int j=0;j<stpair[0].size();j++)
			{
			 d[stpair[0][j].s*S[0]*L[1]+k*S[0]+j]=0;
			 count++;
			}
		}
	int off=nodenum*S[0]*L[1];
	for(int k=0;k<L[2];k++)
		{
		for(int j=0;j<stpair[1].size();j++)
			{
			 d[stpair[1][j].s*S[1]*L[2]+k*S[1]+j+off]=0;
			 count++;
			}
		}
	Size[0]=nodenum*L[1]*S[0];
	Size[1]=nodenum*L[2]*S[1];
	hipMemcpy(dev_d,d,ncount*nodenum*sizeof(int),hipMemcpyHostToDevice);
}
void Bellmanor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,int _nodenum)
{
	nodenum=_nodenum;
	pnodesize=nodenum/(NUT);
	edges=ext.first;
	esigns=ext.second;
	stp=stpair;
	W=WD+1;
	d=new int[nodenum*LY*YE];
	p=new int[nodenum*LY*YE];
	w=new int[edges.size()*LY];
	esignes=new int[edges.size()*LY];
	vector<vector<int>>nein(pnodesize*LY,vector<int>());
	neibn=nein;
	vector<vector<int>>neie(pnodesize,vector<int>());
	vector<vector<int>>rs(pnodesize,vector<int>());
	vector<vector<int>>rw(pnodesize,vector<int>());
	rus=rs;
	ruw=rw;
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			rus[t].push_back(s);
			ruw[t].push_back(i);
			neibn[s].push_back(t);
			neie[s].push_back(i);
		}
	rudu=new int[edges.size()];
	rudw=new int[edges.size()*LY];
	rid=new int[edges.size()];
	int cou1=0;
	int cou2=0;
	int cou3=0;
	mm=new int[pnodesize+1];
	ss=new int[pnodesize+1];
	int du=0;
	for(int i=0;i<pnodesize;i++)
		{
			ss[i]=rus[i].size();
			mm[i]=du;
			du+=rus[i].size();
			for(int j=0;j<rus[i].size();j++)
				rudu[cou1++]=rus[i][j];
			for(int j=0;j<rus[i].size();j++)	
				rid[cou3++]=ruw[i][j];
		}
	for(int k=0;k<LY;k++)
	{
		for(int i=0;i<pnodesize;i++)
		for(int j=0;j<rus[i].size();j++)
			rudw[cou2++]=esigns[k][ruw[i][j]];
	}
	int count=0;
	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_p,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_mm,(pnodesize+1)*sizeof(int));
	hipMalloc((void**)&dev_ss,(pnodesize+1)*sizeof(int));
	hipMalloc((void**)&dev_rudu,edges.size()*sizeof(int));
	hipMalloc((void**)&dev_rudw,edges.size()*LY*sizeof(int));
	hipMalloc((void**)&dev_rid,edges.size()*sizeof(int));
	hipMemcpy(dev_rudu,rudu,edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rudw,rudw,edges.size()*LY*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rid,rid,edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_mm,mm,(pnodesize+1)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_ss,ss,(pnodesize+1)*sizeof(int),hipMemcpyHostToDevice);
};
Bellmanor::Bellmanor():L(PC+1,0),S(PC,0),NF(PC,0),Size(2,0)
{
};
__global__ void bellmandu(int *rudu,int*rudw,int *rid,int *d,int*p,int K,int EE,int PN,int sizeoff,int leveloff,int yel,int ye,int*mm,int *ss)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=yel)return;
	int ly=i/ye+leveloff;
	int nid=blockIdx.y;
	int off=(K-1)*PN*yel+sizeoff;
	int ii=K*PN*yel+nid*yel+i+sizeoff;
	int dm=d[ii];
	int pm=-1;
	for(int k=mm[nid];k<ss[nid]+mm[nid];k++)
		{
			int node=rudu[k];
			if(rudw[k+EE*ly]<0)continue;
			int v=d[off+node*yel+i]+rudw[k+EE*ly];
			if(dm>v)dm=v,pm=rid[k];
		}
	if(d[ii]>dm)
		{
			d[ii]=dm,p[ii]=pm;
		}
}
vector<vector<Rout>> Bellmanor::routalg(int s,int t,int bw)
{
	int kk=1;
	time_t start,end;
	hipStream_t stream0;
	hipStreamCreate(&stream0);
	hipStream_t stream1;
	hipStreamCreate(&stream1);
	dim3 blocks_s1(S[0]*L[1]/512+1,pnodesize);
	dim3 blocks_s2(S[1]*L[2]/512+1,pnodesize);
	int sizeoff=S[0]*L[1]*nodenum;
	for(int i=1;i<WD+1;i++)
	{
		bellmandu<<<blocks_s1,512,0,stream0>>>(dev_rudu,dev_rudw,dev_rid,dev_d,dev_p,i,edges.size(),pnodesize,0,0,S[0]*L[1],S[0],dev_mm,dev_ss);
		bellmandu<<<blocks_s2,512,0,stream0>>>(dev_rudu,dev_rudw,dev_rid,dev_d,dev_p,i,edges.size(),pnodesize,sizeoff,L[1],S[1]*L[2],S[1],dev_mm,dev_ss);
	}
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream0);
	hipMemcpy(d,dev_d,ncount*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(p,dev_p,ncount*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	/*for(int i=39;i<40;i++)
	{
		cout<<"********************************************** "<<i<<endl;
		for(int j=3;j<4;j++)
			{
				cout<<endl;
				for(int k=3;k<4;k++)
					{
						for(int g=33;g<34;g++)
							{
								//cout<<k*S[0]*L[1]*pnodesize+g*S[0]*L[1]+i*S[0]+j<<" ";
								cout<<edges[p[sizeoff+k*S[1]*L[2]*pnodesize+g*S[1]*L[2]+i*S[1]+j]].s<<" ";
								cout<<sizeoff+k*S[1]*L[2]*pnodesize+g*S[1]*L[2]+i*S[1]+j<<endl;
							}
				}
			}
	}*/
	vector<vector<Rout>>result(2,vector<Rout>());
	vector<int>LL(3,0);
	LL=L;
	LL[2]+=LL[1];
	int count=0;
	int offg=0;
	for(int y=1;y<PC+1;y++)
		{
		int leoff=S[y-1]*L[y]*pnodesize;
		int teoff=S[y-1]*L[y];
		for(int k=LL[y-1];k<LL[y];k++)
		{	
			if(y==2)offg=sizeoff;
			int boff=(k-LL[y-1])*S[y-1]+offg;
			for(int l=0;l<stps[y-1].size();l++)
			{	
				int loff=boff+l;
				int s=stps[y-1][l].s;
				vector<int>ters=stps[y-1][l].ters;
				for(int i=0;i<ters.size();i++)
				{
					int id=stps[y-1][l].mmpid[ters[i]];
					int hop=0;
					int tt=ters[i];
					int min=100000;
					int prn=-1;
					for(int v=1;v<W;v++)
						{
						if(d[loff+v*leoff+tt*teoff]<min)
							{	
								min=d[loff+v*leoff+tt*teoff];
								prn=v;
							}
						}
					if(prn<0||min>50000)continue;
					int of=loff+prn*leoff;
					Rout S(s,tt,id,min,of,k);
					result[y-1].push_back(S);
				}
				count++;
			}
		}
		}
	//cout<<"GPU time is : "<<end-start<<endl;
	return result;
};

/*
__global__ void bellmanhigh(int *st,int *te,int *d,int *has,int *w,int E,int N,int size,int *m,int round,int Leveloff,int numoff,int ye,int ly)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*ly));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid],weight=w[eeid];
	if(weight<0)return;
	int off=(i/(E*ly))*N+(eid/E)*N*ye+numoff;
	//if(has[s+off]<round-1)return;
	if(d[s+off]+weight<d[t+off])  
		{
			d[t+off]=weight+d[s+off];
			//has[t+off]=round;
			*m=1;
		}
}*/
/*__global__ void color(int *st,int *te,int *d,int *pre,int *has,int *w,int E,int N,int size,int round,int Leveloff,int numoff,int ye,int ly)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*ly));
	int eeid=eid+Leveloff;
	int s=st[eeid],t=te[eeid],weight=w[eeid];
	if(weight<0)return;
	int off=(i/(E*ly))*N+(eid/E)*N*ye+numoff;
	//if(has[s+off]<round-1)return;
	if(d[s+off]+weight==d[t+off])
		pre[t+off]=s+off;
}*/
/*m1=1;
	*m2=1;
	int round=1;
	cout<<"fuck wx!"<<endl;
	int flag1=0,flag2=0;
	int cc=0;
	while(*m2==1||*m1==1)
	{
		*m2=0,*m1=0;
		hipMemcpyAsync(dev_m2,m2,sizeof(int),hipMemcpyHostToDevice,stream1);
		bellmanhigh<<<size[1]/1024+1,1024,0,stream1>>>(dev_st,dev_te,dev_d,dev_has,dev_w,edges.size(),nodenum,size[1],dev_m2,round,leveloff[1],nodeoff[1],S[1],L[1]);
		hipMemcpyAsync(dev_m1,m1,sizeof(int),hipMemcpyHostToDevice,stream0);
		bellmanhigh<<<size[0]/1024+1,1024,0,stream0>>>(dev_st,dev_te,dev_d,dev_has,dev_w,edges.size(),nodenum,size[0],dev_m2,round,leveloff[0],nodeoff[0],S[0],L[0]);
		color<<<size[1]/1024+1,1024,0,stream1>>>(dev_st,dev_te,dev_d,dev_p,dev_has,dev_w,edges.size(),nodenum,size[1],round,leveloff[1],nodeoff[1],S[1],L[1]);
		hipMemcpyAsync(m2,dev_m2,sizeof(int),hipMemcpyDeviceToHost,stream1);
		color<<<size[0]/1024+1,1024,0,stream0>>>(dev_st,dev_te,dev_d,dev_p,dev_has,dev_w,edges.size(),nodenum,size[0],round,leveloff[0],nodeoff[0],S[0],L[0]);
		hipMemcpyAsync(m1,dev_m1,sizeof(int),hipMemcpyDeviceToHost,stream0);
		hipStreamSynchronize(stream1);
		hipStreamSynchronize(stream0);
	}*/
